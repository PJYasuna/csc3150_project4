#include "hip/hip_runtime.h"
﻿#include "file_system.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


// fcb
// byte 0: 0x40 means read; 0x80means write
// byte 1-20: name
// byte 21-22 address
// byte 23-26 size (4 byte for 1024kb max file)
// byte 27-28 created time
// byte 29-30 modified time
#define PERMISSION 0
#define NAME 1
#define ADDR 21 // 2 byte, 0~32kb, represent the address of bitmap or block
#define SIZE 23
#define C_TIME 27
#define M_TIME 29


__device__ __managed__ int gtime = 0;
__device__ __managed__ u32 free_position = 0;

__device__ int fcb_attribute_addr(FileSystem *fs, int i, int offset);

__device__ void fs_init(FileSystem *fs, uchar *volume, int SUPERBLOCK_SIZE,
							int FCB_SIZE, int FCB_ENTRIES, int VOLUME_SIZE,
							int STORAGE_BLOCK_SIZE, int MAX_FILENAME_SIZE, 
							int MAX_FILE_NUM, int MAX_FILE_SIZE, int FILE_BASE_ADDRESS)
{
  // init variables
  fs->volume = volume;

  // init constants
  fs->SUPERBLOCK_SIZE = SUPERBLOCK_SIZE;
  fs->FCB_SIZE = FCB_SIZE;
  fs->FCB_ENTRIES = FCB_ENTRIES;
  fs->STORAGE_SIZE = VOLUME_SIZE;
  fs->STORAGE_BLOCK_SIZE = STORAGE_BLOCK_SIZE;
  fs->MAX_FILENAME_SIZE = MAX_FILENAME_SIZE;
  fs->MAX_FILE_NUM = MAX_FILE_NUM;
  fs->MAX_FILE_SIZE = MAX_FILE_SIZE;
  fs->FILE_BASE_ADDRESS = FILE_BASE_ADDRESS;

  for (int i = 0; i < 1024; i++) {
    fs->volume[fcb_attribute_addr(fs,i,PERMISSION)] = 0;
  }
}



__device__ int fcb_attribute_addr(FileSystem *fs, int i, int offset) {
  return fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i + offset;
}

__device__ int get_value(FileSystem *fs, int i, int offset) {
  int ans = 0;
  if (offset == M_TIME || offset == ADDR) {
    ans += fs->volume[fcb_attribute_addr(fs, i, offset)] * 256;
    ans += fs->volume[fcb_attribute_addr(fs, i, offset + 1)];
  }
  if (offset == SIZE) {
    ans = fs->volume[fcb_attribute_addr(fs,i,SIZE)] * (256 * 256 * 256);
    ans += fs->volume[fcb_attribute_addr(fs,i,SIZE+1)] * (256 * 256);
    ans += fs->volume[fcb_attribute_addr(fs,i,SIZE+2)] * 256;
    ans += fs->volume[fcb_attribute_addr(fs,i,SIZE+3)];
  }
  return ans;
}

__device__ bool string_compare(FileSystem* fs, int index, char * s2) {
  for (int j = 0; j < 20; j++) {
    if (fs->volume[fcb_attribute_addr(fs,index,NAME) + j] != s2[j]) return false;
    if (s2[j] == '\0') return true;
  }
}

__device__ int block_addr(FileSystem *fs, int i) {
  return get_value(fs,i,ADDR) * fs->STORAGE_BLOCK_SIZE + fs->FILE_BASE_ADDRESS;
}

__device__ void printname(uchar* s) {
  while (*s != '\0') {
		printf("%c", *s);
		s++;
	}
}

__device__ char* get_name(FileSystem *fs, int fp) {
  char ans[20];
	for (int i = 0; i < fs->MAX_FILENAME_SIZE; i++) {
		ans[i] = fs->volume[fcb_attribute_addr(fs,fp,NAME + i)];
		if (fs->volume[fcb_attribute_addr(fs,fp,NAME + i)] == '\0') break; // finish setting name
	}
	return ans;
}

__device__ void update_bitmap(FileSystem *fs, int addr, int res) {
  int byte_addr = addr / 8;
  int bit_addr = addr % 8;
  uchar tmp = 0x1 << bit_addr;
  if (res == 1) {
    fs->volume[byte_addr] |= tmp;
  }
  else {
    tmp = ~tmp;
    fs->volume[byte_addr] &= tmp;
  }
}

__device__ u32 my_write(FileSystem *fs, uchar* input, u32 size, u32 fp) {
  // set address
  fs->volume[fcb_attribute_addr(fs,fp,ADDR)] = free_position / 256;
  fs->volume[fcb_attribute_addr(fs,fp,ADDR+1)] = free_position % 256;
  // set size
  fs->volume[fcb_attribute_addr(fs,fp,SIZE)] = size / (256 * 256 * 256);
  fs->volume[fcb_attribute_addr(fs,fp,SIZE+1)] = size / (256 * 256);
  fs->volume[fcb_attribute_addr(fs,fp,SIZE+2)] = size / 256;
  fs->volume[fcb_attribute_addr(fs,fp,SIZE+3)] = size % 256;

  // set time
  fs->volume[fcb_attribute_addr(fs,fp,M_TIME)] = gtime / 256;
  fs->volume[fcb_attribute_addr(fs,fp,M_TIME+1)] = gtime % 256;

  // write into blocks
  for (int i = 0; i < size; i++) {
    fs->volume[block_addr(fs,fp) + i] = input[i];
  }

  //set bit map
  int block_num = size / fs->STORAGE_BLOCK_SIZE; // 32
  if (size % fs->STORAGE_BLOCK_SIZE != 0) block_num += 1;
  // int start = fcb_attribute_addr(fs,fp,ADDR) * 256;
  // start += fcb_attribute_addr(fs,fp,ADDR + 1);

  int start = get_value(fs,fp,ADDR);

  for (int i = 0; i < block_num; i++) {
    update_bitmap(fs, start + i, 1);
  }


}

__device__ int find_fcb_with_addr(FileSystem *fs, int addr) {
   for (int i = 0; i < 1024; i++) {
        int tmp_addr = get_value(fs,i,ADDR);
        if (tmp_addr == addr) {
            return i;
        }   
    }
    return 1024;
}

__device__ void clean(FileSystem *fs, int fp) {

    int start = get_value(fs,fp,ADDR);
    int size = get_value(fs,fp,SIZE);

    int blocks = size / fs->STORAGE_BLOCK_SIZE; // 32
    if (size % fs->STORAGE_BLOCK_SIZE != 0) blocks += 1;

    // -----------below is compact---------------
  
    // update bitmap
    int current_blocks = free_position - blocks; // 32

    for (int i = 0; i < current_blocks; i++) {
        update_bitmap(fs, i, 1);
    }

    for (int i = current_blocks; i < free_position; i++) {
        update_bitmap(fs, i, 0);
    }

    int start_addr = block_addr(fs, fp);
    for (int i = start; i < current_blocks; i++) {
        for (int j = 0; j < 32; j++) {
          fs->volume[start_addr + i * 32 + j] = fs->volume[start_addr + (i + blocks) * 32 + j];
        }

        int fcb_index = find_fcb_with_addr(fs,i+blocks);
        if (fcb_index != 1024) {
          fs->volume[fcb_attribute_addr(fs,fcb_index,ADDR)] = fs->volume[fcb_attribute_addr(fs,i,ADDR)];
          fs->volume[fcb_attribute_addr(fs,fcb_index,ADDR + 1)] = fs->volume[fcb_attribute_addr(fs,i,ADDR + 1)];
        }

    }

    free_position -= blocks;

}
    



__device__ u32 fs_open(FileSystem *fs, char *s, int op)
{
	/* Implement open operation here */
  gtime++;
  u32 fp;
  int found = 0;

  // found process
  for (int i = 0; i < 1024; i++) {
    if (fs->volume[fcb_attribute_addr(fs,i,PERMISSION)] != 0) { // have exist a file
      // check if the name matches
      if (string_compare(fs, i, s)) {
        found = 1;
        fp = i;
        break;
      }
    }
  }
  // if this file exist
  if (found == 1) {
    // read after write, change 10000000 to 11000000
    if (op == G_READ) {
      fs->volume[fcb_attribute_addr(fs,fp,PERMISSION)] = 0xc0;
    }
    if (op == G_WRITE) {
      clean(fs,fp);
    }
    
    return fp;
  }

  // if not exist
  else {
    if (op == G_READ) printf("No file found for read\n");

    else if (op == G_WRITE) {
      // find empty postion
      int empty = -1;
      for(int i = 0; i < 1024; i++) {
        if(fs->volume[fcb_attribute_addr(fs,i,PERMISSION)] == 0) {
          empty = i;
          break;
        }
      }
      // if don't have empty
      if (empty == -1) printf("Can't open! Too much file\n");

      // if have empty
      else {
        fp = empty;
        fs->volume[fcb_attribute_addr(fs,fp,PERMISSION)] = 0x80;
 
        // set name
        for (int i = 0; i < 20; i++) {
		      fs->volume[fcb_attribute_addr(fs,fp,NAME + i)] = s[i];
		      if (s[i] == '\0') break;
	      }

        //set address
        fs->volume[fcb_attribute_addr(fs,fp,ADDR)] = 0;
        fs->volume[fcb_attribute_addr(fs,fp,ADDR+1)] = 0;

        //set size
        fs->volume[fcb_attribute_addr(fs,fp,SIZE)] = 0;
        fs->volume[fcb_attribute_addr(fs,fp,SIZE+1)] = 0;
        fs->volume[fcb_attribute_addr(fs,fp,SIZE+2)] = 0;
        fs->volume[fcb_attribute_addr(fs,fp,SIZE+3)] = 0;
        
        // set creat time
        fs->volume[fcb_attribute_addr(fs,fp,C_TIME)] = gtime / 256;
        fs->volume[fcb_attribute_addr(fs,fp,C_TIME+1)] = gtime % 256;

        // set modified time
        fs->volume[fcb_attribute_addr(fs,fp,M_TIME)] = gtime / 256;
        fs->volume[fcb_attribute_addr(fs,fp,M_TIME+1)] = gtime % 256;

        return fp;
      }
    }

    // wrong parameter
    else printf("Wrong paramater for operation\n"); 
  }

}


__device__ void fs_read(FileSystem *fs, uchar *output, u32 size, u32 fp)
{
  gtime++;
  if (fs->volume[fcb_attribute_addr(fs,fp,PERMISSION)] & 0x40 != 0x40) {
    printf("Can't read! Don't have read permission.\n");
    return;
  }
  else {
    int address = block_addr(fs,fp);
    for (int i = 0; i < size; i++) {
      output[i] = fs->volume[address + i];
    }
  }
}



__device__ u32 fs_write(FileSystem *fs, uchar* input, u32 size, u32 fp)
{
  gtime++;
  // printf("%d\n", fp);

  if (fs->volume[fcb_attribute_addr(fs,fp,PERMISSION)] & 0x80 != 0x80) {
    printf("Can't write! Don't have write permission.\n");
    return;
  }
  else {
    int blocks = size / fs->STORAGE_BLOCK_SIZE; // 32
    if (size % fs->STORAGE_BLOCK_SIZE != 0) blocks += 1;
    
    // If no enough room
    if (1024 * 1024 - free_position * 32 < size) {
        printf("%d\n", free_position * 32);
        printf("Can't write! the file size exceed the limit\n");
    }
   
    else {
      my_write(fs,input,size,fp);
      free_position += blocks;
    }
  }
  return 0;
}

__device__ void fs_gsys(FileSystem *fs, int op)
{
  gtime++;

  int fp[1024];
  int count = 0;
  for (int i = 0; i < 1024; i++) {
    if (fs->volume[fcb_attribute_addr(fs,i,PERMISSION)] != 0) {
      fp[count] = i;
      count += 1;
    }
  }

	if (op == LS_D) { // sort by modified time
    printf("===sort by modified time===\n");

    for (int i = 0; i < count; i++) {
      for (int j = 0; j < count - 1; j++) {
        if (get_value(fs,fp[j],M_TIME) < get_value(fs,fp[j + 1],M_TIME)) {
          int tmp = fp[j];
          fp[j] = fp[j+1];
          fp[j+1] = tmp;
        }
      }
    }
    for (int i = 0; i < count; i++) {
        printf("%s\n", get_name(fs,fp[i]));
    }
  }

  else if (op == LS_S) {
    printf("===sort by size===\n");
    for (int i = 0; i < count; i++) {
      for (int j = 0; j < count - 1; j++) {
        if (get_value(fs,fp[j],SIZE) < get_value(fs,fp[j + 1],SIZE)) {
          int tmp = fp[j];
          fp[j] = fp[j+1];
          fp[j+1] = tmp;
        }
        else if (get_value(fs,fp[j],SIZE) == get_value(fs,fp[j+1],SIZE) && get_value(fs,fp[j],C_TIME) > get_value(fs,fp[j+1],C_TIME)) {
          int tmp = fp[j];
          fp[j] = fp[j+1];
          fp[j+1] = tmp;
        }
      }
    }

    for (int i = 0; i < count; i++) {
        printf("%s %d\n", get_name(fs,fp[i]), get_value(fs,fp[i],SIZE));
    }


  }

  else printf("Wrong operation!\n");
}

__device__ void fs_gsys(FileSystem *fs, int op, char *s)
{
    gtime++;

    if (op == RM) {
        int fp = -1;
        for(int i = 0; i < 1024; i++) {
            if(string_compare(fs, i, s)) {
            fp = i;
            break;
            }
        }
        if (fp == -1) printf("can not find this file\n");

        else {
            fs->volume[fcb_attribute_addr(fs,fp,PERMISSION)] = 0;

            clean(fs,fp);

            // printf("after clean\n");
            // uchar* name = &fs->volume[fcb_attribute_addr(fs,0,NAME)];
            // printname(name);
            // uchar* name2 = &fs->volume[fcb_attribute_addr(fs,1,NAME)];
            // printname(name2);
        }

    } 
    else printf("Wrong operation!\n");
}
